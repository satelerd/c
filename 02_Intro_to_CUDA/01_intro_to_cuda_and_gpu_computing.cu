/*
Introduction to CUDA and GPU Computing on Azure

What is CUDA?
CUDA (Compute Unified Device Architecture) is a parallel computing platform and API model designed by NVIDIA. 
It allows developers to use NVIDIA GPUs for general purpose processing, significantly accelerating computational tasks by harnessing the power of these GPUs.

Why Use CUDA?
Unlike a CPU that processes tasks sequentially with a few cores, GPUs have thousands of smaller, efficient cores designed for parallel processing. 
This makes GPUs particularly powerful for algorithms that process large data blocks simultaneously, such as in scientific computing, video processing, and deep learning.

Key Concepts in CUDA
- Parallelism: CUDA uses the parallel nature of GPU architecture to perform large-scale computations more efficiently.
- Kernels: In CUDA, a kernel is essentially a function that runs on the GPU. Each kernel operates on different data using multiple threads.
- Threads and Blocks: CUDA executes kernels using a grid of thread blocks. These threads can execute independently but can also synchronize within their block to share data and coordinate execution.

Setting Up a CUDA Project on Azure
Using Azure allows you to access powerful NVIDIA GPUs without needing a physical GPU in your machine. Let's set up your environment to run CUDA programs on Azure.

Prerequisites:
- An Azure account with GPU capabilities (through Microsoft for Startups sponsorship).
- Basic familiarity with Azure's interface.
- Visual Studio Code (VSCode) installed on your local machine.

Steps to Set Up Your Azure GPU Environment:
1. Create a Virtual Machine with a GPU on Azure:
   - Log in to the Azure Portal.
   - Create a new resource and select 'Virtual Machines'.
   - Choose a GPU-enabled VM like the NC Series, which is specifically designed for compute-intensive applications. Select an appropriate region that supports GPU VMs.
   - Configure the VM with your desired settings and create it. Ubuntu is recommended as it is widely supported and compatible with the CUDA Toolkit.

2. Install the CUDA Toolkit on Your VM:
   - Once your VM is set up, connect to it via SSH.
   - Install the CUDA Toolkit by following the commands provided by the NVIDIA CUDA Toolkit installation guide for Linux. This often includes updating your package manager and installing the 'cuda' package.

3. Configure VSCode for Remote Development:
   - Install the Remote Development extension pack in VSCode on your local machine.
   - Connect VSCode to your Azure VM using the Remote-SSH feature, which allows you to edit and run code directly on the VM.

Writing Your First CUDA Program on Azure
Now that your environment is set up, let’s write a simple CUDA program to ensure everything is working properly.

Create and Run a CUDA Program:
- On your Azure VM, create a new file called 'hello.cu' and enter the following CUDA code:
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU ()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");
    helloFromGPU <<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}

/*
Compile the program on the VM using the 'nvcc' compiler from the NVIDIA toolkit:
nvcc hello.cu -o hello

Run the compiled executable:
./hello

You should see "Hello World from CPU!" followed by "Hello World from GPU!" printed multiple times.

Reflection
By setting up and running a CUDA program on Azure, you've taken the first step in leveraging cloud-based GPU computing. 
This approach allows you to develop and test applications that require significant computational power without the need for local hardware. 
As you progress through the course, you'll explore more complex and computationally demanding CUDA programs, fully utilizing the capabilities of GPU computing on Azure.
*/
